#include "hip/hip_runtime.h"
//
// Created by ltc on 2021/11/1.
//
#include "api.cuh"

void cuda::add(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelAdd<<<grid, block>>>(m1, m2, r);
	hipDeviceSynchronize();
}

void cuda::sub(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelSub<<<grid, block>>>(m1, m2, r);
	hipDeviceSynchronize();
}

void cuda::mul(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE/CALCULATE_NUM, BLOCK_SIZE);
	kernelMul<<<grid, block>>>(m1, m2, r);
	hipDeviceSynchronize();
}

void cuda::numMul(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
            double num,
            Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelNumMul<<<grid, block>>>(m1, num, r);
	hipDeviceSynchronize();
}

void cuda::dot(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelDot<<<grid, block>>>(m1, m2, r);
	hipDeviceSynchronize();
}

void cuda::log(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelLog<<<grid, block>>>(m1, r);
	hipDeviceSynchronize();
}

void cuda::maxPool(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
             int channel, int dataRow, int dataCol, int kernelRow, int kernelCol, int stride,
             Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	kernelMaxPool<<<m1.rows(), dataRow*dataCol>>>(m1, channel, dataRow, dataCol, kernelRow, kernelCol, stride, r);
	hipDeviceSynchronize();
}

void cuda::maxPoolBp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
               int channel, int dataRow, int dataCol, int kernelRow, int kernelCol, int stride,
               Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	kernelMaxPoolBp<<<r.rows(), dataRow*dataCol>>>(m1, channel, dataRow, dataCol, kernelRow, kernelCol, stride, r);
	hipDeviceSynchronize();
}

void cuda::exp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelExp<<<grid, block>>>(m1, r);
	hipDeviceSynchronize();
}

void cuda::pow(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
         double num,
         Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelPow<<<grid, block>>>(m1, num, r);
	hipDeviceSynchronize();
}

void cuda::convToImg(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
               int num,
               Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((m1.rows()-1)/BLOCK_SIZE+1, (m1.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelConvToImg<<<grid, block>>>(m1, num, r);
	hipDeviceSynchronize();
}

void cuda::convToImgBp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                 int num,
                 Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelConvToImgBp<<<grid, block>>>(m1, num, r);
	hipDeviceSynchronize();
}

void cuda::imgToConv(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
               int channel, int dataRow, int dataCol, int kernelRow, int kernelCol, int stride,
               Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	kernelImgToConv<<<m1.rows(), dataRow*dataCol>>>(m1, channel, dataRow, dataCol, kernelRow, kernelCol, stride, r);
	hipDeviceSynchronize();
}

void cuda::imgToConvBp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                 int channel, int dataRow, int dataCol, int kernelRow, int kernelCol, int stride,
                 Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	kernelImgToConvBp<<<r.rows(), dataRow*dataCol>>>(m1, channel, dataRow, dataCol, kernelRow, kernelCol, stride, r);
	hipDeviceSynchronize();
}

void cuda::setValue(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
              double num) {
	dim3 grid((m1.rows()-1)/BLOCK_SIZE+1, (m1.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelSetValue<<<grid, block>>>(m1, num);
	hipDeviceSynchronize();
}

void cuda::setValue(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
              int row, int col, double value) {
	kernelSetValue<<<1, 1>>>(m1, row, col, value);
	hipDeviceSynchronize();
}

double cuda::getValue(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                int row, int col) {
	double* r;
	double result;
	hipMalloc((void**)&r, sizeof(double));
	kernelGetValue<<<1, 1>>>(m1, row, col, r);
	hipDeviceSynchronize();
	hipMemcpy(&result, r, sizeof(double), hipMemcpyDeviceToHost);
	hipFree(r);
	return result;
}

void cuda::info(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1) {
	kernelInfo<<<1, 1>>>(m1);
	hipDeviceSynchronize();
}

void cuda::transpose(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
               Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((r.rows()-1)/BLOCK_SIZE+1, (r.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelTranspose<<<grid, block>>>(m1, r);
	hipDeviceSynchronize();
}

double cuda::max(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1) {
	dim3 grid((m1.rows()-1)/BLOCK_SIZE+1, (m1.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	double* r;
	hipMalloc((void**)&r, grid.x*grid.y*sizeof(double));
	kernelMax<<<grid, block>>>(m1, r);
	hipDeviceSynchronize();
	double tmp[grid.x*grid.y];
	hipMemcpy(tmp, r, grid.x*grid.y*sizeof(double), hipMemcpyDeviceToHost);
	hipFree(r);
	double result = DBL_MIN;
	for (int i = 0; i < grid.x*grid.y; ++i) {
		result = std::max(result, tmp[i]);
	}
	return result;
}

double cuda::min(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1) {
	dim3 grid((m1.rows()-1)/BLOCK_SIZE+1, (m1.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	double* r;
	hipMalloc((void**)&r, grid.x*grid.y*sizeof(double));
	kernelMin<<<grid, block>>>(m1, r);
	hipDeviceSynchronize();
	double tmp[grid.x*grid.y];
	hipMemcpy(tmp, r, grid.x*grid.y*sizeof(double), hipMemcpyDeviceToHost);
	hipFree(r);
	double result = DBL_MAX;
	for (int i = 0; i < grid.x*grid.y; ++i) {
		result = std::min(result, tmp[i]);
	}
	return result;
}

void cuda::relu(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1) {
	dim3 grid((m1.rows()-1)/BLOCK_SIZE+1, (m1.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelRelu<<<grid, block>>>(m1);
	hipDeviceSynchronize();
}

void cuda::reluBp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
            Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
            Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	dim3 grid((m1.rows()-1)/BLOCK_SIZE+1, (m1.cols()-1)/BLOCK_SIZE+1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	kernelReluBp<<<grid, block>>>(m1, m2, r);
	hipDeviceSynchronize();
}
