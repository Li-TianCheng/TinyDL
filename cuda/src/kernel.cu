#include "hip/hip_runtime.h"
//
// Created by ltc on 2021/11/1.
//
#include "kernel.cuh"

__global__ void kernelAdd(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) = m1(row, col) + m2(row, col);
	}
}

__global__ void kernelSub(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) = m1(row, col) - m2(row, col);
	}
}

__global__ void kernelMul(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int deltaX = blockDim.x * gridDim.x;
	const unsigned int deltaY = blockDim.y * gridDim.y;
	const unsigned int blockSize = BLOCK_SIZE/CALCULATE_NUM;
	__shared__ double sharedM1[CALCULATE_NUM][blockSize][blockSize];
	__shared__ double sharedM2[CALCULATE_NUM][blockSize][blockSize];
	double re[CALCULATE_NUM][CALCULATE_NUM] = {0};
	unsigned int num = (m1.cols()-1) / blockSize + 1;
	for (int i = 0; i < num; ++i) {
		unsigned int y1 = threadIdx.y + i * blockSize;
		unsigned int x2 = threadIdx.x + i * blockSize;
		for (int j = 0; j < CALCULATE_NUM; ++j) {
			unsigned int x1 = row + j * deltaX;
			unsigned int y2 = col + j * deltaY;
			sharedM1[j][threadIdx.x][threadIdx.y] = x1 < m1.rows() && y1 < m1.cols() ? m1(x1, y1) : 0;
			sharedM2[j][threadIdx.x][threadIdx.y] = x2 < m2.rows() && y2 < m2.cols() ? m2(x2, y2) : 0;
		}
		__syncthreads();
		for (int j = 0; j < blockSize; ++j) {
			for (int k = 0; k < CALCULATE_NUM; ++k) {
				for (int l = 0; l < CALCULATE_NUM; ++l) {
					re[k][l] += sharedM1[k][threadIdx.x][j] * sharedM2[l][j][threadIdx.y];
				}
			}
		}
		__syncthreads();
	}
	for (int i = 0; i < CALCULATE_NUM; ++i) {
		unsigned int x = row + i * deltaX;
		for (int j = 0; j < CALCULATE_NUM; ++j) {
			unsigned int y = col + j * deltaY;
			if (x < r.rows() && y < r.cols()) {
				r(x, y) = re[i][j];
			}
		}
	}
}

__global__ void kernelNumMul(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                             double num,
                             Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) = m1(row, col) * num;
	}
}

__global__ void kernelDot(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) = m1(row, col) * m2(row, col);
	}
}

__global__ void kernelLog(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) = log(m1(row, col));
	}
}

__global__ void kernelMaxPool(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                              int dataRow, int dataCol, int kernelRow, int kernelCol, int stride,
                              Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int i = blockIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.x;
	if (j < ((dataRow-kernelRow)/stride+1)*((dataCol-kernelCol)/stride+1)) {
		int kx = j / ((dataCol-kernelCol)/stride+1);
		int ky = j % ((dataCol-kernelCol)/stride+1);
		int x0 = kx*stride;
		int y0 = ky*stride;
		double value = DBL_MIN;
		for (int m = 0; m < kernelRow; ++m) {
			for (int n = 0; n < kernelCol; ++n) {
				int x = x0 + m;
				int y = y0 + n;
				int idx = x*dataCol+y+blockIdx.z*dataRow*dataCol;
				value = max(value, m1(i, idx));
			}
		}
		r(i, j+blockIdx.z*((dataRow-kernelRow)/stride+1)*((dataCol-kernelCol)/stride+1)) = value;
	}
}

__global__ void kernelMaxPoolBp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                                Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
                                int dataRow, int dataCol, int kernelRow, int kernelCol, int stride,
                                Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m3,
                                Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int i = blockIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.x;
	if (j < dataRow*dataCol) {
		int x = j / dataCol;
		int y = j % dataCol;
		double value = r(i, j+blockIdx.z*dataRow*dataCol);
		double max = m1(i, j+blockIdx.z*dataRow*dataCol);
		for (int k = 0; k < kernelRow*kernelCol; ++k) {
			int kx = k / kernelCol;
			int ky = k % kernelCol;
			int kx0 = x - kx;
			int ky0 = y - ky;
			if (kx0 >= 0 && ky0 >= 0 && kx0 <= dataRow-kernelRow && ky0 <= dataCol-kernelCol && kx0 % stride == 0 && ky0 % stride == 0) {
				int n = kx0 / stride * ((dataRow-kernelRow)/stride+1) + ky0 / stride;
				if (m2(i, n+blockIdx.z*((dataRow-kernelRow)/stride+1)*((dataCol-kernelCol)/stride+1)) == max) {
					value += m3(i, n+blockIdx.z*((dataRow-kernelRow)/stride+1)*((dataCol-kernelCol)/stride+1));
				}
			}
		}
		r(i, j+blockIdx.z*dataRow*dataCol) = value;
	}
}

__global__ void kernelExp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) = exp(m1(row, col));
	}
}

__global__ void kernelPow(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          double num,
                          Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) = pow(m1(row, col), num);
	}
}

__global__ void kernelConvToImg(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                                int num,
                                Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < m1.rows() && col < m1.cols()) {
		r(row/num, col*num+row%num) = m1(row, col);
	}
}

__global__ void kernelConvToImgBp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                                  int num,
                                  Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) += m1(row/num, col*num+row%num);
	}
}

__global__ void kernelImgToConv(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                                int dataRow, int dataCol, int kernelRow, int kernelCol, int stride,
                                Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int i = blockIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.x;
	if (j < dataRow*dataCol) {
		int x = j / dataCol;
		int y = j % dataCol;
		double value = m1(i, j+blockIdx.z*dataRow*dataCol);
		for (int k = 0; k < kernelRow*kernelCol; ++k) {
			int kx = k / kernelCol;
			int ky = k % kernelCol;
			int kx0 = x - kx;
			int ky0 = y - ky;
			if (kx0 >= 0 && ky0 >= 0 && kx0 <= dataRow-kernelRow && ky0 <= dataCol-kernelCol && kx0 % stride == 0 && ky0 % stride == 0) {
				int n = kx0 / stride * ((dataRow - kernelRow) / stride + 1) + ky0 / stride;
				r(i*((dataRow-kernelRow)/stride+1)*((dataCol-kernelCol)/stride+1)+n, k+blockIdx.z*kernelRow*kernelCol) = value;
			}
		}
	}
}

__global__ void kernelImgToConvBp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                                  int dataRow, int dataCol, int kernelRow, int kernelCol, int stride,
                                  Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int i = blockIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.x;
	if (j < dataRow*dataCol) {
		int x = j / dataCol;
		int y = j % dataCol;
		double value = r(i, j+blockIdx.z*dataCol*dataRow);
		for (int k = 0; k < kernelRow*kernelCol; ++k) {
			int kx = k / kernelCol;
			int ky = k % kernelCol;
			int kx0 = x - kx;
			int ky0 = y - ky;
			if (kx0 >= 0 && ky0 >= 0 && kx0 <= dataRow-kernelRow && ky0 <= dataCol-kernelCol && kx0 % stride == 0 && ky0 % stride == 0) {
				int n = kx0 / stride * ((dataRow-kernelRow)/stride+1) + ky0 / stride;
				value += m1(blockIdx.x*((dataRow-kernelRow)/stride+1)*((dataCol-kernelCol)/stride+1)+n, k+blockIdx.z*kernelRow*kernelCol);
			}
		}
		r(i, j+blockIdx.z*dataCol*dataRow) = value;
	}
}

__global__ void kernelSetValue(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                               double num) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < m1.rows() && col < m1.cols()) {
		if (num == -1) {
			if (row == col) {
				m1(row, col) = 1;
			} else {
				m1(row, col) = 0;
			}
		} else {
			m1(row, col) = num;
		}
	}
}

__global__ void kernelGetValue(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                               int row, int col, double* r) {
	*r = m1(row, col);
}

__global__ void kernelInfo(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1) {
	for (int i = 0; i < m1.rows(); ++i) {
		for (int j = 0; j < m1.cols(); ++j) {
			printf("%f ", m1(i, j));
		}
		printf("\n");
	}
}

__global__ void kernelTranspose(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                                Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < r.rows() && col < r.cols()) {
		r(row, col) = m1(col, row);
	}
}

__global__ void kernelMax(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          double* r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	__shared__ double mem[BLOCK_SIZE*BLOCK_SIZE];
	double tmp = row < m1.rows() && col < m1.cols() ? m1(row, col) : DBL_MIN;
	int idx = threadIdx.x*BLOCK_SIZE + threadIdx.y;
	mem[idx] = tmp;
	__syncthreads();
#pragma unroll 9
	for (int stride = BLOCK_SIZE*BLOCK_SIZE/2; stride > 0; stride >>= 1) {
		__syncthreads();
		if (idx < stride) {
			if (mem[idx] < mem[idx+stride]) {
				mem[idx] = mem[idx+stride];
			}
		}
	}
	if (idx == 0) {
		r[blockIdx.x*gridDim.y+blockIdx.y] = mem[0];
	}
}

__global__ void kernelMin(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                          double* r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	__shared__ double mem[BLOCK_SIZE*BLOCK_SIZE];
	double tmp = row < m1.rows() && col < m1.cols() ? m1(row, col) : DBL_MAX;
	int idx = threadIdx.x*BLOCK_SIZE + threadIdx.y;
	mem[idx] = tmp;
	__syncthreads();
#pragma unroll 9
	for (int stride = BLOCK_SIZE*BLOCK_SIZE/2; stride > 0; stride >>= 1) {
		__syncthreads();
		if (idx < stride) {
			if (mem[idx] > mem[idx+stride]) {
				mem[idx] = mem[idx+stride];
			}
		}
	}
	if (idx == 0) {
		r[blockIdx.x*gridDim.y+blockIdx.y] = mem[0];
	}
}

__global__ void kernelRelu(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < m1.rows() && col < m1.cols()) {
		if (m1(row, col) < 0) {
			m1(row, col) = 0;
		}
	}
}

__global__ void kernelReluBp(Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m1,
                             Map<Matrix<double, Dynamic, Dynamic, RowMajor>> m2,
                             Map<Matrix<double, Dynamic, Dynamic, RowMajor>> r) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < m1.rows() && col < m1.cols()) {
		if (m1(row, col) != 0) {
			r(row, col) += m2(row, col);
		}
	}
}
